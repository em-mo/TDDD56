#include "hip/hip_runtime.h"
// Matrix addition, CPU version
// gcc matrix_cpu.c -o matrix_cpu -std=c99

#include <stdio.h>

__global__ 
void simple(float *a, float *b, float *c, int N) 
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	int index = idx + idy*N;

	if(index < N*N)
		c[index] = a[index] + b[index];
}

void add_matrix(float *a, float *b, float *c, int N)
{
	int index;
	
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			index = i + j*N;
			c[index] = a[index] + b[index];
		}
}

int main()
{ 
	const int N = 128;
	float *a = new float[N*N];
	float *b = new float[N*N];
	float *c = new float[N*N];
	float *c2 = new float[N*N];

	float* ad;
	float* bd;
	float* cd;

	int size = N*N* sizeof(float);
	int gridX = 4;
	int gridY = 4;

	hipEvent_t start_event;
	hipEvent_t end_event;
	float theTime;

	hipMalloc( (void**)&ad, size );
	hipMalloc( (void**)&bd, size );
	hipMalloc( (void**)&cd, size );

	hipEventCreate(&start_event);
	hipEventCreate(&end_event);

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			a[i+j*N] = 10 + i;
			b[i+j*N] = (float)j / N;
		}
	}

	hipMemcpy( ad, a, size, hipMemcpyHostToDevice ); 
	hipMemcpy( bd, b, size, hipMemcpyHostToDevice ); 

	dim3 dimBlock((N+1)/gridX, (N+1)/gridY);
	dim3 dimGrid( gridX, gridY );

	hipEventRecord(start_event, 0);
	hipEventSynchronize(start_event);
	
	simple<<<dimGrid, dimBlock>>>(ad, bd, cd, N);

	hipDeviceSynchronize();
  	hipEventRecord(end_event, 0);
	hipEventSynchronize(end_event);
  	
	hipEventElapsedTime(&theTime, start_event, end_event);

	hipMemcpy( c, cd, size, hipMemcpyDeviceToHost ); 	

	/*for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%0.2f ", c[i+j*N]);
		}
		printf("\n");
	}*/

	add_matrix(a, b, c, N);

	for(int i = 0; i < N*N; i++)
	{
		if(c[i] != c2[i]){
			printf("olika! %f!=%f\n", c[i], c2[i]);
			break;
		}
	}

	printf("time: %f ms\n", theTime);
}
